#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/hip/hip_vector_types.h"

#define BLOCK_X 16
#define BLOCK_Y 16
#define CHECK_CUDA(x) AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define DEBUG
texture<float, hipTextureType1D, hipReadModeElementType> sinoTexture;

__device__ inline void _atomicAdd(float *address, float val){
      int i_val = __float_as_int(val);
      int tmp0 = 0, tmp1;
      while((tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0){
        tmp0 = tmp1;
        i_val = __float_as_int(val + __int_as_float(tmp1));
      }
}

__global__ void differentiableKernel(float* error, float* volume, float* projectVector, const uint2 volumeSize, const float2 volumeCenter, const float detectorCenter){
    uint2 volumeIdx = make_uint2(blockIdx.x*blockDim.x+threadIdx.x, blockIdx.y*blockDim.y+threadIdx.y);
    if (volumeIdx.x >= volumeSize.x || volumeIdx.y >= volumeSize.y) return;

    float2 volumeCoordination = make_float2(volumeIdx) + (float2)volumeCenter;
    float3 point = make_float3(volumeCoordination, 1);
    float volumeError = volume[volumeIdx.y * volumeSize.x + volumeIdx.x];
    float3 projectVectorSp1 = make_float3(projectVector[0], projectVector[1], projectVector[2]);
    float3 projectVectorSp2 = make_float3(projectVector[3], projectVector[4], projectVector[5]);
    float u = dot(point, projectVectorSp1);
    float v = dot(point, projectVectorSp2);
    float gsino = tex1Dfetch(sinoTexture, __fdividef(u, v) - detectorCenter);
    float3 errorSp1 = point * gsino * volumeError;
    float3 errorSp2 = point * -u * gsino * volumeError;
    if(fabs(v)>1e-5){
      errorSp1 /= v;
      errorSp2 /= (v*v);
    }

    #if defined DEBUG
    if(volumeIdx.x == 256 && volumeIdx.y == 256){
      printf("point(%f,%f,%f)\n", point.x, point.y, point.z);
      printf("u=%f v=%f\n",u,v);
      printf("w=%f\n", __fdividef(u, v) - detectorCenter);
      printf("grad(D)=%f\n", gsino);
      printf("errorSp1(%f,%f,%f)\n", errorSp1.x, errorSp1.y, errorSp1.z);
      printf("errorSp2(%f,%f,%f)\n", errorSp2.x, errorSp2.y, errorSp2.z);
      printf("idx=%d\n", volumeIdx.y * volumeSize.x + volumeIdx.x);
      printf("volume error=%f\n", volumeError);
    }
    #endif

    _atomicAdd(error, errorSp1.x);
    _atomicAdd(error+1, errorSp1.y);
    _atomicAdd(error+2, errorSp1.z);
    _atomicAdd(error+3, errorSp2.x);
    _atomicAdd(error+4, errorSp2.y);
    _atomicAdd(error+5, errorSp2.z);
}

torch::Tensor differentiableFanFlatGradient(torch::Tensor sino, torch::Tensor volume, torch::Tensor projectVector){
    CHECK_INPUT(sino);
    CHECK_INPUT(volume);
    CHECK_INPUT(projectVector);
    AT_ASSERTM(projectVector.size(0) == 2, "project vector's shape 0 must be 2");
    AT_ASSERTM(projectVector.size(1) == 3, "project vector's shape 1 must be 3");

    auto out = torch::zeros({2,3}).to(volume.device());
    float* outPtr = out.data<float>();
    float* sinoPtr = sino.data<float>();
    float* volumePtr = volume.data<float>();
    float* projectVectorPtr = projectVector.data<float>();

    const uint2 volumeSize = make_uint2(512, 512);
    const float2 volumeCenter = make_float2(volumeSize) / -2.0;
    const uint detectorSize = 900;
    const float detectorCenter = detectorSize / -2.0;

    sinoTexture.filterMode = hipFilterModeLinear;
    sinoTexture.normalized = false;
    hipBindTexture(NULL, sinoTexture, sinoPtr, detectorSize*sizeof(float));
    const dim3 blockSize = dim3(BLOCK_X, BLOCK_Y);
    const dim3 gridSize = dim3(volumeSize.x/BLOCK_X, volumeSize.y/BLOCK_Y);
    differentiableKernel<<<gridSize, blockSize>>>(outPtr, volumePtr, projectVectorPtr, volumeSize, volumeCenter, detectorCenter);
    hipUnbindTexture(sinoTexture);

    return out;
}