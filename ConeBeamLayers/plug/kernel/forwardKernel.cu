#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/hip/hip_vector_types.h"

#define BLOCK_X 16
#define BLOCK_Y 16
#define PI 3.14159265359
#define CHECK_CUDA(x) AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// 存储体块的纹理内存
texture<float, hipTextureType3D, hipReadModeElementType> volumeTexture;

__global__ void forwardKernel(float* sino, const uint3 volumeSize, const float3 volumeCenter, const uint2 detectorSize, const float2 detectorCenter, const float* projectVector, const uint index){
    // 像素驱动，此核代表一个探测器像素
    uint3 detectorIdx = make_uint3(blockIdx.x*blockDim.x+threadIdx.x, blockIdx.y*blockDim.y+threadIdx.y, blockIdx.z*blockDim.z+threadIdx.z);
    if (detectorIdx.x >= detectorSize.x || detectorIdx.y >= detectorSize.y){
        return;
    }

    float detectorX = detectorIdx.x + detectorCenter.x;
    float detectorY = detectorIdx.y + detectorCenter.y;

    float3 sourcePosition = make_float3(projectVector[index*12], projectVector[index*12+1], -projectVector[index*12+2]);
    float3 detectorPosition = make_float3(projectVector[index*12+3], projectVector[index*12+4], -projectVector[index*12+5]);
    float3 v = make_float3(projectVector[index*12+6], projectVector[index*12+7], projectVector[index*12+8]);
    float3 u = make_float3(projectVector[index*12+9], projectVector[index*12+10], projectVector[index*12+11]);

    // 计算当前角度下的中心射线方向向量与探测器像素的位置坐标
    float3 detectorPixel = detectorPosition + (0.5f+detectorX) *u + (0.5f+detectorY) * v ;
    // 计算得到像素射线方向和起始点
    float3 rayVector = normalize(detectorPixel - sourcePosition);

    // 计算范围并累加
    float pixel = 0.0f;
    float alpha0, alpha1;
    float rayVectorDomainDim=fmax(fabs(rayVector.x),fmax(fabs(rayVector.z),fabs(rayVector.y)));
    if (fabs(rayVector.x) == rayVectorDomainDim){
        float volume_min_edge_point = volumeCenter.x;
        float volume_max_edge_point = volumeSize.x + volumeCenter.x;
        alpha0 = (volume_min_edge_point - sourcePosition.x) / rayVector.x;
        alpha1 = (volume_max_edge_point - sourcePosition.x) / rayVector.x;
    }
    else if(fabs(rayVector.y) == rayVectorDomainDim){
        float volume_min_edge_point = volumeCenter.y;
        float volume_max_edge_point = volumeSize.y + volumeCenter.y;
        alpha0 = (volume_min_edge_point - sourcePosition.y) / rayVector.y;
        alpha1 = (volume_max_edge_point - sourcePosition.y) / rayVector.y;
    }
    else {
        float volume_min_edge_point = volumeCenter.z;
        float volume_max_edge_point = volumeSize.z + volumeCenter.z;
        alpha0 = (volume_min_edge_point - sourcePosition.z) / rayVector.z;
        alpha1 = (volume_max_edge_point - sourcePosition.z) / rayVector.z;
    }
    float min_alpha = fmin(alpha0, alpha1) - 3;
    float max_alpha = fmax(alpha0, alpha1) + 3;
    float px, py, pz;

    while (min_alpha<max_alpha)
    {
        px = sourcePosition.x + min_alpha * rayVector.x;
        py = sourcePosition.y + min_alpha * rayVector.y;
        pz = sourcePosition.z + min_alpha * rayVector.z;
        px -= volumeCenter.x;
        py -= volumeCenter.y;
        pz -= volumeCenter.z;
        pixel += tex3D(volumeTexture, px + 0.5f, py + 0.5f, pz + 0.5f);
        min_alpha ++;
    }
    unsigned sinogramIdx = index * detectorSize.x * detectorSize.y + detectorIdx.y * detectorSize.x + detectorIdx.x;
    sino[sinogramIdx] = pixel;
}

torch::Tensor forward(torch::Tensor volume, torch::Tensor _volumeSize, torch::Tensor _detectorSize, torch::Tensor projectVector, const long device){
    CHECK_INPUT(volume);
    CHECK_INPUT(_volumeSize);
    AT_ASSERTM(_volumeSize.size(0) == 3, "volume size's length must be 3");
    CHECK_INPUT(_detectorSize);
    AT_ASSERTM(_detectorSize.size(0) == 2, "detector size's length must be 2");
    CHECK_INPUT(projectVector);
    AT_ASSERTM(projectVector.size(1) == 12, "project vector's shape must be [angle's number, 12]");

    int angles = projectVector.size(0);
    auto out = torch::zeros({volume.size(0), 1, angles, _detectorSize[1].item<int>(), _detectorSize[0].item<int>()}).to(volume.device());
    float* outPtr = out.data<float>();
    float* volumePtr = volume.data<float>();

    // 初始化纹理
    hipSetDevice(device);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    volumeTexture.addressMode[0] = hipAddressModeBorder;
    volumeTexture.addressMode[1] = hipAddressModeBorder;
    volumeTexture.addressMode[2] = hipAddressModeBorder;
    volumeTexture.filterMode = hipFilterModeLinear;
    volumeTexture.normalized = false;

    // 体块和探测器的大小位置向量化
    uint3 volumeSize = make_uint3(_volumeSize[0].item<int>(), _volumeSize[1].item<int>(), _volumeSize[2].item<int>());
    float3 volumeCenter = make_float3(volumeSize) / -2.0;
    uint2 detectorSize = make_uint2(_detectorSize[0].item<int>(), _detectorSize[1].item<int>());
    float2 detectorCenter = make_float2(detectorSize) / -2.0;

    for(int batch = 0;batch < volume.size(0); batch++){
        float* volumePtrPitch = volumePtr + volumeSize.x * volumeSize.y * volumeSize.z * batch;
        float* outPtrPitch = outPtr + angles * detectorSize.x * detectorSize.y * batch;

        // 绑定纹理
        hipExtent m_extent = make_hipExtent(volumeSize.x, volumeSize.y, volumeSize.z);
        hipArray *volumeArray;
        hipMalloc3DArray(&volumeArray, &channelDesc, m_extent);
        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr = make_hipPitchedPtr((void*)volumePtrPitch, volumeSize.x*sizeof(float), volumeSize.x, volumeSize.y);
        copyParams.dstArray = volumeArray;
        copyParams.kind = hipMemcpyDeviceToDevice;
        copyParams.extent = m_extent;
        hipMemcpy3D(&copyParams);
        hipBindTextureToArray(volumeTexture, volumeArray, channelDesc);

        // 以角度为单位做探测器像素驱动的正投影
        const dim3 blockSize = dim3(BLOCK_X, BLOCK_Y, 1);
        const dim3 gridSize = dim3(detectorSize.x / BLOCK_X + 1, detectorSize.y / BLOCK_Y + 1, 1);
        for (int angle = 0; angle < angles; angle++){
           forwardKernel<<<gridSize, blockSize>>>(outPtrPitch, volumeSize, volumeCenter, detectorSize, detectorCenter, (float*)projectVector.data<float>(), angle);
        }

      // 解绑纹理
      hipUnbindTexture(volumeTexture);
      hipFreeArray(volumeArray);
    }
    return out;
}